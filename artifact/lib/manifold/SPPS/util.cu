#include "hip/hip_runtime.h"

#include "config.h"
#include <stdio.h>


// �Ծ���A������һ��y����y+alpha*x֮�󣬷������B�Ķ�Ӧ����
__global__ void _column_wise_add(psFloat* B, psFloat alpha, psFloat* A, int r, int c,  psFloat beta, psFloat* x)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	
	if (i >= r || j >= c)
		return;

	B[j*r+i] = alpha*A[j*r+i]+beta*x[i];
}

void column_wise_add(psFloat* B, psFloat alpha, psFloat* A, int r, int c,psFloat beta, psFloat* x)
{
	dim3 dimBlock(16,32);
	dim3 dimGrid((r+dimBlock.x-1)/dimBlock.x, (c+dimBlock.y-1)/dimBlock.y);

	_column_wise_add<<<dimGrid,dimBlock>>>(B,alpha, A, r, c, beta, x);
}

// �Ծ���A�е�j��y����y = x[j]*y֮�󣬷������B�Ķ�Ӧ����
__global__ void _column_wise_scal(psFloat* B, psFloat alpha, psFloat* A, int r, int c, psFloat* x)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	
	if (i >= r || j >= c)
		return;

	B[j*r+i] = alpha* A[j*r+i]*x[j];
}

void column_wise_scal(psFloat* B, psFloat alpha, psFloat* A, int r, int c, psFloat* x)
{
	dim3 dimBlock(16,32);
	dim3 dimGrid((r+dimBlock.x-1)/dimBlock.x, (c+dimBlock.y-1)/dimBlock.y);
	_column_wise_scal<<<dimGrid,dimBlock>>>(B, alpha, A, r, c,  x);
}

// C[i,j] = A[i,j]*B[i,j]
__global__ void _element_wise_scal(psFloat* C, psFloat alpha, psFloat* A, psFloat * B, int r, int c)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	
	if (i >= r || j >= c)
		return;

	C[j*r+i] = alpha * A[j*r+i]*B[j*r+i];
}

void element_wise_scal(psFloat* C,psFloat alpha, psFloat* A, psFloat * B, int r, int c)
{
	dim3 dimBlock(16,32);
	dim3 dimGrid((r+dimBlock.x-1)/dimBlock.x, (c+dimBlock.y-1)/dimBlock.y);
	_element_wise_scal<<<dimGrid,dimBlock>>>(C, alpha, A, B, r, c);
}

// �Ծ���A�е�j��y����y = y/x[j]֮�󣬷������B�Ķ�Ӧ����
__global__ void _column_wise_normal(psFloat* B, psFloat* A, int r, int c, psFloat* x)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	
	if (i >= r || j >= c)
		return;

	B[j*r+i] = A[j*r+i]/x[j];
}

void column_wise_normal(psFloat* B,psFloat* A, int r, int c, psFloat* x)
{
	dim3 dimBlock(16,32);
	dim3 dimGrid((r+dimBlock.x-1)/dimBlock.x, (c+dimBlock.y-1)/dimBlock.y);
	_column_wise_normal<<<dimGrid,dimBlock>>>(B, A, r, c,  x);
}

__global__ void _column_wise_sum(psFloat* n, psFloat* A, int r, int c)
{
	int bid = blockIdx.x;
	int tid = threadIdx.x;
	
	if (tid >= r || bid >=c)
		return;

	extern __shared__ psFloat s_data[];
	extern __shared__ psFloat curLen;

	s_data[tid] = A[bid*c+tid];

	__syncthreads();

	curLen = r;

	for (int i = (curLen+1)/2; i > 0 ; i = (curLen+1)/2 )
	{
		if(tid < i && tid+i < curLen)
		{
			s_data[tid]   += s_data[tid+i];
		}
		curLen = i;
		__syncthreads();
	}
	
	if (tid == 0)
		n[bid] = s_data[0];

}

void column_wise_sum(psFloat* n, psFloat* A, int r, int c)
{
	_column_wise_sum<<<c,r,r>>>(n, A, r, c);
}


__global__ void _column_wise_norm2(psFloat* n, psFloat* A, int r, int c)
{
	int bid = blockIdx.x ;
	int tid = threadIdx.x;



	// Add the overflow checking!
	
	if (tid >= r || bid >= c  )
		return;
		
	extern __shared__ psFloat s_data[]; 
	extern __shared__ psFloat curLen;

	
	s_data[tid] = A[bid*r+tid]*A[bid*r+tid];

	__syncthreads();

	curLen = r;

	for (int i = (curLen+1)/2; i > 0 ; i = (curLen+1)/2 )
	{
		if(tid < i && tid+i < curLen)
		{
			s_data[tid]   += s_data[tid+i];
		}
		curLen = i;
		__syncthreads();
	}
	
	if (tid == 0)
		n[bid] = sqrt(s_data[0]);

}


void column_wise_norm2(psFloat* n, psFloat* A, int r, int c)
{
	_column_wise_norm2<<<c,r,r>>>(n, A, r, c);
}


__global__ void _column_wise_dot(psFloat* n, psFloat* A, psFloat * B, int r, int c)
{
	int bid = blockIdx.x ;
	int tid = threadIdx.x;



	// Add the overflow checking!
	
	if (tid >= r || bid >= c  )
		return;
		
	extern __shared__ psFloat s_data[]; 
	
	s_data[tid] = A[bid*r+tid]*B[bid*r+tid];

	__syncthreads();

	curLen = r;

	for (int i = (curLen+1)/2; i > 0 ; i = (curLen+1)/2 )
	{
		if(tid < i && tid+i < curLen)
		{
			s_data[tid]   += s_data[tid+i];
		}
		curLen = i;
		__syncthreads();
	}
	
	if (tid == 0)
		n[bid] = s_data[0];

}


void column_wise_dot(psFloat* n, psFloat* A, psFloat * B, int r, int c)
{
	_column_wise_dot<<<c,r,r>>>(n, A, B, r, c);
}


__global__ void _column_wise_squared_norm2(psFloat* n, psFloat* A, int r, int c)
{
	int bid = blockIdx.x ;
	int tid = threadIdx.x;



	// Add the overflow checking!
	
	if (tid >= r || bid >= c  )
		return;
		
	extern __shared__ psFloat s_data[]; 
	extern __shared__ psFloat curLen;

	
	s_data[tid] = A[bid*r+tid]*A[bid*r+tid];

	__syncthreads();

	curLen = r;

	for (int i = (curLen+1)/2; i > 0 ; i = (curLen+1)/2 )
	{
		if(tid < i && tid+i < curLen)
		{
			s_data[tid]   += s_data[tid+i];
		}
		curLen = i;
		__syncthreads();
	}
	
	if (tid == 0)
		n[bid] = s_data[0];

}


void column_wise_squared_norm2(psFloat* n, psFloat* A, int r, int c)
{
	_column_wise_squared_norm2<<<c,r,r>>>(n, A, r, c);
}

__global__ void _column_wise_squared_distance_mm(psFloat* n, psFloat* A, psFloat * B, int r, int c)
{
	int bid = blockIdx.x ;
	int tid = threadIdx.x;

	// Add the overflow checking!
	
	if (tid >= r || bid >= c  )
		return;
		
	extern __shared__ psFloat s_data[]; 
	extern __shared__ psFloat curLen;

	
	s_data[tid] = (A[bid*r+tid]-B[bid*r+tid])*(A[bid*r+tid]-B[bid*r+tid]);

	curLen = r;

	for (int i = (curLen+1)/2; i > 0 ; i = (curLen+1)/2 )
	{
		if(tid < i && tid+i < curLen)
		{
			s_data[tid]   += s_data[tid+i];
		}
		curLen = i;
		__syncthreads();
	}
	
	if (tid == 0)
		n[bid] = sqrt(s_data[0]);
}

void column_wise_squared_distance_mm(psFloat* n, psFloat* A, psFloat* B, int r, int c)
{
	_column_wise_squared_distance_mm<<<c,r,r>>>(n, A, B, r, c);
}




__global__ void _column_wise_distance_mv(psFloat* dis, psFloat* A, int r, int c, psFloat* x)
{

	int bid = blockIdx.x;
	int tid = threadIdx.x;

	if (tid >= r ||   bid >= c)
		return;
		
	extern __shared__ psFloat s_data[];
	extern __shared__ psFloat curLen;


	s_data[tid] = (A[bid*r+tid]-x[tid])*(A[bid*r+tid]-x[tid]);

	__syncthreads();

	curLen = r;

	for (int i = (curLen+1)/2; i > 0 ; i = (curLen+1)/2 )
	{
		if(tid < i && tid+i < curLen)
		{
			s_data[tid]   += s_data[tid+i];
		}
		curLen = i;
		__syncthreads();
	}
	
	if (tid == 0)
		dis[bid] = sqrt(s_data[0]);
}

void column_wise_distance_mv(psFloat* n, psFloat* A, int r, int c,psFloat* x)
{
	_column_wise_distance_mv<<<c,r,r>>>(n, A, r, c,x);
}




__global__ void _fill(psFloat* a, unsigned int N, unsigned int inc, psFloat val)
{

	int i = blockIdx.x * blockDim.x + threadIdx.x;
	
	int location = i * inc;
	
	if (location >= N)
		return;
		
	a[location] = val;
	
} 


void fill(psFloat* a, unsigned int N, unsigned int inc, psFloat val)
{
	int numGrid = (N + 512 -1)/512;
	_fill<<<numGrid,512>>>(a, N,inc,val);
}


#define BLOCK_DIM                      16
/**
  * Computes the distance between two matrix A (reference points) and
  * B (query points) containing respectively wA and wB points.
  *
  * @param A     pointer on the matrix A
  * @param wA    width of the matrix A = number of points in A
  * @param pA    pitch of matrix A given in number of columns
  * @param B     pointer on the matrix B
  * @param wB    width of the matrix B = number of points in B
  * @param pB    pitch of matrix B given in number of columns
  * @param dim   dimension of points = height of matrices A and B
  * @param AB    pointer on the matrix containing the wA*wB distances computed
  */
__global__ void cuComputeDistanceGlobal( psFloat* A, int wA, int pA, psFloat* B, int wB, int pB, int dim,  psFloat* AB){

	// Declaration of the shared memory arrays As and Bs used to store the sub-matrix of A and B
	__shared__ psFloat shared_A[BLOCK_DIM][BLOCK_DIM];
	__shared__ psFloat shared_B[BLOCK_DIM][BLOCK_DIM];
    
    // Sub-matrix of A (begin, step, end) and Sub-matrix of B (begin, step)
    __shared__ int begin_A;
    __shared__ int begin_B;
    __shared__ int step_A;
    __shared__ int step_B;
    __shared__ int end_A;
	
    // Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;
	
	// Other variables
	psFloat tmp;
    psFloat ssd = 0;
	
    // Loop parameters
    begin_A = BLOCK_DIM * blockIdx.y;
    begin_B = BLOCK_DIM * blockIdx.x;
    step_A  = BLOCK_DIM * pA;
    step_B  = BLOCK_DIM * pB;
    end_A   = begin_A + (dim-1) * pA;
    
    // Conditions
	int cond0 = (begin_A + tx < wA); // used to write in shared memory
    int cond1 = (begin_B + tx < wB); // used to write in shared memory & to computations and to write in output matrix
    int cond2 = (begin_A + ty < wA); // used to computations and to write in output matrix
    
    // Loop over all the sub-matrices of A and B required to compute the block sub-matrix
    for (int a = begin_A, b = begin_B; a <= end_A; a += step_A, b += step_B) {
        
        // Load the matrices from device memory to shared memory; each thread loads one element of each matrix
        if (a/pA + ty < dim){
            shared_A[ty][tx] = (cond0)? A[a + pA * ty + tx] : 0;
            shared_B[ty][tx] = (cond1)? B[b + pB * ty + tx] : 0;
        }
        else{
            shared_A[ty][tx] = 0;
            shared_B[ty][tx] = 0;
        }
        
        // Synchronize to make sure the matrices are loaded
        __syncthreads();
        
        // Compute the difference between the two matrixes; each thread computes one element of the block sub-matrix
        if (cond2 && cond1){
            for (int k = 0; k < BLOCK_DIM; ++k){
				tmp = shared_A[k][ty] - shared_B[k][tx];
                ssd += tmp*tmp;
			}
        }
        
        // Synchronize to make sure that the preceding computation is done before loading two new sub-matrices of A and B in the next iteration
        __syncthreads();
    }
    
    // Write the block sub-matrix to device memory; each thread writes one element
    if (cond2 && cond1)
        AB[ (begin_A + ty) * pB + begin_B + tx ] = ssd;
}


void pairwise_squared_distance( psFloat* RQ, psFloat* ref, int wR, int pR, psFloat* query, int wQ, int pQ, int dim )
{
	dim3 g(wQ/BLOCK_DIM, wR/BLOCK_DIM, 1);
    dim3 t(BLOCK_DIM, BLOCK_DIM, 1);

    if (wQ	%	BLOCK_DIM != 0) g.x += 1;
    if (wR	%	BLOCK_DIM != 0) g.y += 1;

	cuComputeDistanceGlobal<<<g,t>>>(ref, wR, pR, query, wQ, pQ, dim,  RQ);

}


__global__ void columnwise_min_index(int * index, psFloat * A, int r, int c)
{
	int bid = blockIdx.x;
	int tid = threadIdx.x;
	
	if (tid >= r || bid >=c)
		return;

	extern __shared__ psFloat s_data[];
	extern __shared__ psFloat curLen;
	extern __shared__ psFloat s_index[];


	s_data[tid] = A[bid*c+tid];
	index[tid] = tid;

	__syncthreads();


	curLen = r;

	for (int i = (curLen+1)/2; i > 0 ; i = (curLen+1)/2 )
	{
		if(tid < i && tid+i < curLen)
		{
			if (s_data[tid] > s_data[tid+i])
			{
				s_data[tid] =  s_data[tid+i];
				s_index[tid] = s_index[tid+i];
			}
		}
		curLen = i;
		__syncthreads();
	}

	
	if (tid == 0)
		index[bid] = s_index[0];

}

void columnwise_min_index(psFloat* n, psFloat* A, int r, int c)
{
	columnwise_min_index<<<c,r,r>>>(n, A, r, c);
}

__global__ void _copy_indexed_columns(psFloat * X, int * index, int nX, psFloat * A, int r, int c)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	
	if (i >= r || j >= nX)
		return;

	X[j*r+i] = A[index[j]*r+i];
}

void copy_indexed_columns(psFloat * X, int * index, int nX, psFloat * A, int r, int c)
{
	dim3 dimBlock(16,32);
	dim3 dimGrid((r+dimBlock.x-1)/dimBlock.x, (c+dimBlock.y-1)/dimBlock.y);

	_copy_indexed_columns<<<dimGrid,dimBlock>>>(X,index, nX, A, r, c);
}
