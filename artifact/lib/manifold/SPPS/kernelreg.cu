#include "hip/hip_runtime.h"

#include <math.h>
#include <hipblas.h>
#include <cutil.h>

#include "config.h"
#include "cumath.h"
#include "util.h"

#include "spps.h"

namespace KernelRegressionManifold
{

	psFloat * _Distance,* _B, * _P, * _Q;  //nS * nX matrix
	psFloat * _temp_nS_nX_1, * _temp_nS_nX_2, * _temp_nS_nX_3;             // nS * nX matrix
	psFloat * _MX ; // Dim * nX   matrix
	psFloat * _temp_nX_1;  // nX  array
	psFloat * _temp_dim_nX_1;  // dim * nX matrix



	void slps_regkernel(psFloat * kernelVal, psFloat * kernelD, psFloat* kernelD2, psFloat * S, int nS,  psFloat * X, int nX, int dim,  psFloat * sigma, KernelType kernel, CalculationType calType)
	{

		// static * distance;
		pairwise_distance(S, nS, nS*sizeof(psFloat), X, nX*sizeof(psFloat), nX, dim, _Distance);

	#ifdef _DEBUG_DETAIL
		display("the resulting normDiffV matrix is:", _Distance, nS ,  nX);
	#endif

		if (kernel == Gaussian)
		{
			rowwise_gausian_kernel( kernelVal, kernelD, kernelD2, _Distance, nS, nX, sigma, calType);

		}
		else if(kernel == Quadratic)
		{
			rowwise_quadratic_kernel( kernelVal, kernelD, kernelD2,_Distance, nS, nX, sigma, calType);

		}

	#ifdef _DEBUG_DETAIL
		display("the resulting kernelVal matrix is:", kernelVal,nS ,  nX);
	#endif
		
	}




	void slps_kernelregbasis(psFloat * B, psFloat * P, psFloat * Q, psFloat * S, int nS,  psFloat * X, int nX, int dim,  psFloat * sigma, KernelType kernel, CalculationType calType)
	{
		
		// calculate the kernel value and kernel differences
		// kernelVal, kernelD, kernelD2  ������ b, p, q ����
		slps_regkernel(B,P,Q, S,  nS,  X, nX, dim, sigma,kernel, calType);

		psFloat * _sumval = _temp_nX_1;

		column_wise_sum(_sumval, B, nS, nX);
		column_wise_normal(B,B,nS,nX, _sumval);
		
	#ifdef _DEBUG_DETAIL
		display("the resulting b matrix is:", B,nS ,  nX);
	#endif
		
		if (calType == Eval)
			return;
				
		// Calculate p
		column_wise_normal(P,P,nS,nX, _sumval);
		cublasFscal (nS*nX, -2, P, 1);
	#ifdef _DEBUG_DETAIL
		display("the resulting p matrix is:", P, nS ,  nX); 
	#endif
		// Calculate q
		column_wise_normal(Q,Q,nS,nX, _sumval);
		cublasFscal (nS*nX, 4, Q, 1);
	#ifdef _DEBUG_DETAIL
		display("the resulting q matrix is:", Q, nS ,  nX); 
	#endif


	}

	// Calculate object value (mX) of X throught the mapping constructed via Kernel regression from S to T
	void slps_map(psFloat * mX, psFloat * X, int nX, psFloat * T, int Dim, psFloat * S, int nS, int dim,    psFloat * sigma, KernelType kernel, CalculationType calType)
	{
		slps_kernelregbasis(_B, _P, _Q,  S, nS,  X, nX,  dim, sigma, kernel, calType);
		// Compute mx
		// ps.mx = ps.T*ps.b;
		cublasFgemm('N','N',Dim,  nX, nS, 1, T, Dim, _B, nS, 0, mX, Dim);			
	}

	void slps_dist(psFloat * dist, psFloat * JDX,  psFloat * X, int nX, psFloat * Y,  psFloat * T, int Dim, psFloat * S, int nS,  int dim,  psFloat * sigma, KernelType kernel, CalculationType calType)
	{

		slps_map(_MX, X, nX, T, Dim, S, nS,  dim, sigma, kernel, calType);
		
		// tempD = y - ps.mx;
		// dist = norm(tempD);
		// MX =  MX - Y
		cublasFaxpy(Dim*nX, -1, Y, 1, _MX, 1);

		if (calType == Eval || dist != 0)
		{
			column_wise_squared_norm2(dist, _MX, Dim, nX);
			
		}
		if (calType == Eval)
			return;
		
		//Jdx_i = 2*Jmx_i*(mx_i-y_i) = 2*Jbx*T'*(mx_i-y_i) = 2*diffVecs_i*(diag(p_i)-p_i*b_i')T'*(mx_i-y_i)
		// Jdx_i = 2*[(S-repmat(x_i,1,nS))*(diag(p_i)*T'*(mx_i-y_i)-p_i*b_i'*T'*(mx_i-y_i))]
		// _temp_nS_nX_i = T'*(mx_i-y_i);
		cublasFgemm('T','N',nS,nX,Dim, 1, T, Dim, _MX , Dim, 0, _temp_nS_nX_1, nS);
		// _temp_nX_i = b_i'*T'*(mx_i-y_i);
		column_wise_dot(_temp_nX_1, _B, _temp_nS_nX_1 , nS , nX);
		// temp_nS_nX2_i = p_i*b_i'*T'*(mx_i-y_i) 
		column_wise_scal(_temp_nS_nX_2, 1, _P, nS, nX, _temp_nX_1);
		
		// _temp_nS_nX3_i = diag(p_i)*T'*(mx_i-y_i)
		element_wise_scal(_temp_nS_nX_3, 1, _P , _temp_nS_nX_1,nS, nX);
		
		// _temp_nS_nX3_i = (diag(p_i)*T'*(mx_i-y_i)-p_i*b_i'*T'*(mx_i-y_i));
		cublasFaxpy(nS*nX, -1, _temp_nS_nX_2, 1, _temp_nS_nX_3, 1);
		
		// JDX = 2*S*_temp_nS_nX_3;
		cublasFgemm('N','N',dim,nX,nS, 2, S, dim, _temp_nS_nX_3 ,nS, 0, JDX, dim);
		
		// _temp_dim_nX_i = rempmat(x_i,1,ns)*_temp_nS_nX3_i = sum(_temp_nS_nX3_i)*x_i;
		column_wise_sum(_temp_nX_1, _temp_nS_nX_3 ,nS , nX);
		column_wise_scal(_temp_dim_nX_1,2, X, dim, nX, _temp_nX_1);
		
		cublasFaxpy(dim*nX, -1, _temp_dim_nX_1, 1, JDX, 1);
		
	}


	namespace _OptimHelper
	{

		psFloat * Y;
		int nY;
		psFloat * T;
		int Dim; 
		psFloat * S;
		int nS;
		int dim;
		psFloat * Sigma;
		KernelType kernel;
		// �Ż���������,������ĳ��y���������ͶӰָ��
		// �������������ĳ��y��S(x)�ľ���
		// ʹ��ǰ���뽫y����������y�С�
		void slps_dist_help(psFloat * Dist, psFloat* X )
		{

		//	cublasFcopy(s.d, x, 1, x,1);

			slps_dist(Dist, 0,  X, nY,  Y,  T, Dim,  S, nS,  dim,  Sigma, kernel, Eval);

		//	static int i = 0;
			
		//	i ++;
			
		//	if (i % 10 == 0)
		#ifdef _DEBUG_DETAIL
//			display("computing the object value at point:", _x, 1 , s.d);
		#endif

		}

		// �Ż���������,������ĳ��y���������ͶӰָ��
		// �������������ĳ��y��S(x)�ľ����x�ĵ���
		// ʹ��ǰ���뽫y����������y�У�����slps_dist_help��֮ǰ�����ű����ù���
		void slps_dist_Jacobbi_help(psFloat*  _Jdx, psFloat* _x )
		{

			slps_dist(0, JDX,  X, nY,X, nY,  Y,  T, Dim,  S, nS,  dim,  Sigma, kernel, Jacobbi);

		#ifdef _DEBUG_DETAIL	
			//display("computing the Jacobbi matrix at point:", _x, 1 , s.d);
			//display("the resulting Jacobbi matrix is:", _Jdx, 1 , s.d);
		#endif
			
		//	cublasFcopy(s.d, Jdx, 1, Jdx,1);
		}
	}

	void slps_project(psFloat * X, psFloat* Y, int nY,  bool withInitVal, psFloat* X0,  psFloat * T, int Dim, psFloat * S, int nS, int dim, psFloat * Sigma, KernelType kernel)
	{
		/**
		* Ѱ�ҵ���Ӧ�ĳ�ʼ��x
		*/
		_OptimHelper::Y = Y;
		_OptimHelper::nY = nY;
		_OptimHelper::T = T;
		_OptimHelper::Dim = Dim;
		_OptimHelper::S = S;
		_OptimHelper::nS = nS;
		_OptimHelper::dim = dim;
		_OptimHelper::Sigma = Sigma;
		_OptimHelper::kernel = kernel;
		
		if (!withInitVal)
		{
			pairwise_squared_distance( _Distance, S, nS, 0, X , nX , 0, dim);

			// Ѱ��tempM����Сֵ��λ�á�
			columnwise_min_index(_temp_nX_1, _Distance , nS, nX);
			copy_indexed_columns(X, _temp_nX_1, nX, S , dim, nS);
		}
		else
		{
			cublasFcopy(dim*nX, X0, 1, X,1);
		}

	#ifdef _DEBUG_DETAIL	
		display("The initial point is:", _x, 1 , s.d);
	#endif
		int iter;
		psFloat * fret = _temp_nX_1;
		psFloat ftol = 1e-6;
		
		// the resulting x is the projection index.
		batch_frprmn(nX, dim, X, ftol, &iter, fret, &slps_dist_help,&slps_dist_Jacobbi_help);
//		frprmn(s.d, _x, ftol, iter, fret, &slps_dist_help,&slps_dist_Jacobbi_help);

	}

	void kernelreg_init(int dim, int nS, int nX, int Dim )
	{

	//	psFloat * _Distance,* _B, * _P, * _Q;  //nS * nX matrix
	//psFloat * _temp_nS_nX_1, * _temp_nS_nX_2, * _temp_nS_nX_3;             // nS * nX matrix
	//psFloat * _MX ; // Dim * nX   matrix
	//psFloat * _temp_nX_1;  // nX  array
	//psFloat * _temp_dim_nX_1;  // dim * nX matrix

		cublasAlloc(nS*nX, sizeof(psFloat), (void**)& _Distance);
		cublasAlloc(nS*nX, sizeof(psFloat), (void**)& _B);
		cublasAlloc(nS*nX, sizeof(psFloat), (void**)& _P);
		cublasAlloc(nS*nX, sizeof(psFloat), (void**)& _Q);
		cublasAlloc(nS*nX, sizeof(psFloat), (void**)& _temp_nS_nX_1);
		cublasAlloc(nS*nX, sizeof(psFloat), (void**)& _temp_nS_nX_2);
		cublasAlloc(nS*nX, sizeof(psFloat), (void**)& _temp_nS_nX_3);

		cublasAlloc(Dim*nX, sizeof(int), (void**)& _MX);

		cublasAlloc(nX, sizeof(int), (void**)& _temp_nX_1);
		cublasAlloc(dim*nX, sizeof(int), (void**)& _temp_dim_nX_1);

	}

	void kernelreg_final(int dim, int nS, int nX, int Dim )
	{
		cublasFree( _Distance);
		cublasFree( _B);
		cublasFree( _P);
		cublasFree( _Q);
		cublasFree( _temp_nS_nX_1);
		cublasFree( _temp_nS_nX_2);
		cublasFree( _temp_nS_nX_3);

		cublasFree( _MX);

		cublasFree( _temp_nX_1);
		cublasFree( _temp_dim_nX_1);
	}
}
