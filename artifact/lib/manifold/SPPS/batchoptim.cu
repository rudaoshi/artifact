#include "hip/hip_runtime.h"

#include "config.h"
#include <math.h>
#include <hipblas.h>
#include <float.h>
#include "util.h"



__device__ void swap(psFloat& a, psFloat& b)
{
	psFloat c = b;
	b = a;
	a = c;
}


__device__ void shft3(psFloat&a, psFloat&b, psFloat&c, const psFloat d)
{
	a=b;
	b=c;
	c=d;
}

__device__ psFloat sign(const psFloat&a, const psFloat&b)
{
	return b >= 0 ? (a >= 0 ? a : -a) : (a >= 0 ? -a : a);
}

__device__ psFloat GOLD=1.618034,GLIMIT=100.0,TINY=1.0e-20,ZEPS=DBL_EPSILON*1.0e-3, CGOLD=0.3819660,EPS=1.0e-18;

__device__ int ncom, dcom;

psFloat * _Finish_mnbrak;
psFloat * _Finish_brent;
psFloat * _Finish_frprmn;

int * _Step_mnbrak;

psFloat *_temp_dcom_ncom_1;

psFloat * _u, * _fu, * _x, * _fx, * _v, *_fv, * _w, *_fw, *_a,* _b;

__global__ void mnbrak_pre_step( psFloat * finish, psFloat * ax, psFloat * bx, psFloat *cx, psFloat *fa, psFloat *fb, psFloat *fc)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	
	if ( i >= ncom) return;
	
	if (fb[i] > fa[i]) {
		swap(ax[i],bx[i]);
		swap(fb[i],fa[i]);
	}
	cx[i]=bx[i]+GOLD*(bx[i]-ax[i]);

	finish[i] = 0;
	
}


__global__ void mnbrak_step1(psFloat * finish, int * step, psFloat * u, psFloat * ax, psFloat * bx, psFloat *cx, psFloat *fa, psFloat *fb, psFloat *fc)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	
	if ( i >= ncom) return;
	
	if (fb[i] <= fc[i])
	{
		finish[i] = 1;
		return;
	}
	
	psFloat  ulim,r,q;
	
	r=(bx[i]-ax[i])*(fb[i]-fc[i]);
	q=(bx[i]-cx[i])*(fb[i]-fa[i]);
	u[i]=bx[i]-((bx[i]-cx[i])*q-(bx[i]-ax[i])*r)/ (2.0*sign(max(fabs(q-r),TINY),q-r));
	ulim=bx[i]+GLIMIT*(cx[i]-bx[i]);
	if ((bx[i]-u[i])*(u[i]-cx[i]) > 0.0) 
	{
		step[i] = 0;
	} 
	else if ((cx[i]-u[i])*(u[i]-ulim) > 0.0) 
	{
		step[i] = 1;
	}
	else if ((u[i]-ulim)*(ulim-cx[i]) >= 0.0) 
	{
		u[i] = ulim;
		step[i] = 2;
	} 
	else 
	{
		u[i]=cx[i]+GOLD*(cx[i]-bx[i]);
		step[i] = 3;
	}
}

__global__ void mnbrak_step2( psFloat * finish, int * step, psFloat * u,  psFloat * ax, psFloat * bx, psFloat *cx, psFloat *fa, psFloat *fb, psFloat *fc, psFloat * fu)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	
	if ( i >= ncom) return;
	
	if (finish[i]) return;
	
	if (step[i] == 0)
	{
		if (fu[i] < fc[i]) {
			ax[i]=bx[i];
			bx[i]=u[i];
			fa[i]=fb[i];
			fb[i]=fu[i];
			finish[i] = 1;
			return;
		} else if (fu[i] > fb[i]) {
			cx[i]=u[i];
			fc[i]=fu[i];
			finish[i] = 1;
			return;
		}
		u[i]=cx[i]+GOLD*(cx[i]-bx[i]);
	}
	else if (step[i] == 1)
	{
		if (fu < fc) {
			shft3(bx[i],cx[i],u[i],cx[i]+GOLD*(cx[i]-bx[i]));
//			shft3(fb,fc,fu,func(u));
			step[i] = 11;
		}
	}
}

__global__ void mnbrak_step3( psFloat * finish, int * step, psFloat * u,  psFloat * ax, psFloat * bx, psFloat *cx, psFloat *fa, psFloat *fb, psFloat *fc, psFloat * fu)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	
	if ( i >= ncom) return;
	
	if (finish[i]) return;
	
    if (step[i] == 11)
	{
		fb[i] = fc[i] = fu[i];
	}
	
	shft3(ax[i],bx[i],cx[i],u[i]);
	shft3(fa[i],fb[i],fc[i],fu[i]);
	
	if (fb[i] > fc[i]) 
	{
		finish[i] = 1;
	}
	
	
}



// mnbrak���ûƽ���ʺͶ��β�ֵȷ�������ļ�Сֵ�����������
void batch_mnbrak(  psFloat * ax, psFloat * bx, psFloat *cx, psFloat *fa, psFloat *fb, psFloat *fc, void func(psFloat *, psFloat*))
{
	func(fa,ax);
	func(fb,bx);

	int numGrid = (ncom + 512 -1)/512;

	mnbrak_pre_step<<<numGrid,512>>>(  _Finish_mnbrak, ax,  bx, cx, fa, fb, fc);
	func(fc,cx);
	
	while (cublasFasum(ncom, _Finish_mnbrak, 1) <= ncom)
	{
		mnbrak_step1<<<numGrid,512>>>(_Finish_mnbrak, _Step_mnbrak, _u,  ax,  bx, cx, fa, fb, fc);
		func(_fu,_u);
		mnbrak_step2<<<numGrid,512>>>( _Finish_mnbrak,  _Step_mnbrak,  _u,   ax,  bx, cx, fa, fb, fc, _fu);
		func(_fu,_u);
		mnbrak_step3<<<numGrid,512>>>( _Finish_mnbrak, _Step_mnbrak,  _u,  ax,  bx, cx, fa, fb, fc,  _fu);
	}
	
	
}


__global__ void brent_pre_step(psFloat * finish,  psFloat * a, psFloat * b, psFloat * x, psFloat * w, psFloat * v,  psFloat * ax, psFloat * bx, psFloat *cx)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	
	if ( i >= ncom) return;
	
	a[i] = (ax[i] < cx[i] ? ax[i] : cx[i]);
	b[i] = (ax[i] > cx[i] ? ax[i] : cx[i]);
	
	x[i] = w[i] = v[i] = bx[i];

	finish[i] = 0;
}

__global__ void brent_step1( psFloat * finish, psFloat * xmin, psFloat * u,   psFloat * a, psFloat * b, psFloat * x, psFloat * w, psFloat * v, psFloat * fx, psFloat * fv, psFloat * fw, psFloat tol )
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	
	if ( i >= ncom || finish[i] == 1) return;
	
	psFloat xm = 0.5*(a[i] + b[i]);
	
	psFloat p,q,r, e,d, etemp;
	psFloat tol1, tol2;
		
	tol2 = 2.0*(tol1 = tol*fabs(x[i])+ZEPS);
	e = 0.0; d = 0.0;

	if (fabs(x[i]-xm) <= (tol2-0.5*(b[i]-a[i]))) {
		xmin[i] = x[i];
		finish[i] = 1;
		return;
	}
	
	if (fabs(e) > tol1) {
		r=(x[i]-w[i])*(fx[i]-fv[i]);
		q=(x[i]-v[i])*(fx[i]-fw[i]);
		p=(x[i]-v[i])*q-(x[i]-w[i])*r;
		q=2.0*(q-r);
		if (q > 0.0) p = -p;
		q=fabs(q);
		etemp=e;
		e=d;
		if (fabs(p) >= fabs(0.5*q*etemp) || p <= q*(a[i]-x[i]) || p >= q*(b[i]-x[i]))
			d=CGOLD*(e=(x[i] >= xm ? a[i]-x[i] : b[i]-x[i]));
		else {
			d=p/q;
			u[i]=x[i]+d;
			if (u[i]-a[i] < tol2 || b[i]-u[i] < tol2)
				d=sign(tol1,xm-x[i]);
		}
	} else {
		d=CGOLD*(e=(x[i] >= xm ?  a[i]-x[i] : b[i]-x[i]));
	}
	u[i] = (fabs(d) >= tol1 ? x[i]+d : x[i]+sign(tol1,d));
}


__global__ void brent_step2( psFloat * finish,  psFloat * u, psFloat * a, psFloat * b, psFloat * x, psFloat * w, psFloat * v, psFloat * fx, psFloat * fv, psFloat * fw, psFloat * fu)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	
	if ( i >= ncom || finish[i] ) return;
	
	if (fu[i] <= fx[i]) {
		if (u[i] >= x[i]) a[i]=x[i]; else b[i]=x[i];
		shft3(v[i],w[i],x[i],u[i]);
		shft3(fv[i],fw[i],fx[i],fu[i]);
	} else {
		if (u[i] < x[i]) a[i]=u[i]; else b[i]=u[i];
		if (fu[i] <= fw[i] || w[i] == x[i]) {
			v[i]=w[i];
			w[i]=u[i];
			fv[i]=fw[i];
			fw[i]=fu[i];
		} else if (fu[i] <= fv[i] || v[i] == x[i] || v[i] == w[i]) {
			v[i]=u[i];
			fv[i]=fu[i];
		}
	}

}

// ��Сֵ���һά��������
void batch_brent(psFloat * fx, psFloat * xmin, psFloat * ax, psFloat * bx, psFloat * cx, void f(psFloat *, psFloat*), psFloat tol)
{
	const int ITMAX=100;
	int iter;


	//a=(ax < cx ? ax : cx);
	//b=(ax > cx ? ax : cx);
	//x=w=v=bx;

	int numGrid = (ncom + 512 -1)/512;

	brent_pre_step<<<numGrid, 512>>>( _Finish_brent,_a,  _b,  _x,  _w,  _v,  ax,  bx, cx);
	f(fx, _x);
	cublasFcopy(ncom,_fx, 1,_fv,1);
	cublasFcopy(ncom,_fx, 1,_fw,1);
	for (iter=0;iter<ITMAX && cublasFasum(ncom,_Finish_brent,1) < ncom - 0.1 ;iter++) {
		brent_step1<<<numGrid,512>>>( _Finish_brent,  xmin,  _u,   _a,  _b, _x,  _w,  _v,  fx,  _fv,  _fw, tol );
		f(_fu,_u);
		brent_step2<<<numGrid,512>>>( _Finish_brent, _u, _a,  _b,  _x,  _w,  _v,  fx,  _fv,  _fw, _fu  );
	}
//	nrerror("Too many iterations in brent");
	cublasFcopy(ncom,_x, 1,xmin,1);
}

/**
* 
*/


void (*nrfunc)(psFloat*, psFloat*);
psFloat * xt;
psFloat * pcom_p;
psFloat * xicom_p;

void f1dim(psFloat * fx,  psFloat * x)
{
	int j;

	psFloat* pcom = pcom_p, * xicom = xicom_p;

	cublasFcopy(ncom*dcom,pcom, 1,xt,1);
//	cublasFaxpy(ncom,x,xicom, 1,xt,1);
	column_wise_scal(_temp_dcom_ncom_1, 1, xicom , dcom, ncom, x);
	cublasFaxpy(ncom*dcom,1,_temp_dcom_ncom_1, 1,xt,1);
	//for (j=0;j<ncom;j++)
	//	xt[j]=pcom[j]+x*xicom[j];
	nrfunc(fx, xt);
}


psFloat * _ax, * _bx, * _xx , * _fxx, * _fa, * _fb, * _xmin;
void batch_linmin(psFloat* p, psFloat* xi, psFloat* fret, void func(psFloat *, psFloat*))
{
	int j;
	const psFloat TOL=1.0e-8;

	nrfunc=func;

	cublasFcopy(ncom*dcom,p, 1,pcom_p,1);
	cublasFcopy(ncom*dcom,xi, 1,xicom_p,1);
	//Vec_DP &pcom=*pcom_p,&xicom=*xicom_p;
	//for (j=0;j<n;j++) {
	//	pcom[j]=p[j];
	//	xicom[j]=xi[j];
	//}
	
	hipMemset(_ax,0,ncom*sizeof(psFloat));
	fill(_xx,ncom,1,1.0);

	batch_mnbrak( _ax,_xx,_bx,_fa,_fxx,_fb,f1dim);
	batch_brent(fret, _xmin, _ax,_xx,_bx,f1dim,TOL);

//	cublasFscal(ncom, xmin,xi,1);
//	cublasFaxpy(ncom,1,xi,1,p,1);
	//cublasFcopy(ncom*dcom,xim, 1,xi,1);
	column_wise_scal(xi, 1, xi , dcom, ncom, _xmin);
	cublasFaxpy(ncom*dcom,1,p, 1,xi,1);
	//for (j=0;j<n;j++) {
	//	xi[j] *= xmin;
	//	p[j] += xi[j];
	//}
	//delete xicom_p;
	//delete pcom_p;
}

/**
* �����ݶȷ��������Ż���
*/

__global__ void batch_frprmn_step1( psFloat * finish, psFloat * fret, psFloat * fp, psFloat ftol)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	
	if ( i >= ncom || finish[i] == 1) return;
	
	if (2.0*fabs(fret[i]-fp[i]) <= ftol*(fabs(fret[i])+fabs(fp[i])+EPS))
		finish[i] = 1;

}



__global__ void batch_frprmn_step2( psFloat * finish, psFloat * gg, psFloat * dgg, psFloat * xi, psFloat * g)
{
	int bid = blockIdx.x ;
	int tid = threadIdx.x;

	// Add the overflow checking!
	
	if (tid >= dcom || bid >= ncom || finish[bid] == 1 )
		return;
		
	extern __shared__ psFloat gg_data[]; 
	extern __shared__ psFloat dgg_data[]; 
	
	gg_data[tid] = g[bid*dcom+tid]*g[bid*dcom+tid];
	dgg_data[tid] = (xi[bid*dcom+tid]+g[bid*dcom+tid])*xi[bid*dcom+tid];

	__syncthreads();

	for (int i = (dcom+1)/2; i > 0 ; i /=2)
	{
		if(tid < i && tid+i < dcom)
		{
			gg_data[tid] = gg_data[tid] + gg_data[tid+i];
			dgg_data[tid] = dgg_data[tid] + dgg_data[tid+i];
		}
		__syncthreads();
	}
	
	if (tid == 0)
	{
		gg[bid] = gg_data[0];
		dgg[bid] = dgg_data[0];
		
		if (dgg[bid] == 0.0)
		{
			finish[bid] = 1;
			return;
		}	
		
	}
}

__global__ void batch_frprmn_step3( psFloat * finish, psFloat * xi, psFloat * g, psFloat * h, psFloat * gg, psFloat * dgg )
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	
	if (i >= dcom || j >= ncom)
		return;
		
	psFloat gam = dgg[j]/gg[j];

	g[j*dcom+i] = -xi[j*dcom+i];
	xi[j*dcom+i] = h[j*dcom+i] = g[j*dcom+i] + gam*h[j*dcom+i];
}



psFloat * _gg, * _fp, * _dgg, * _g, * _xi, * _h;

void batch_frprmn(int n, int d, psFloat* p, const psFloat ftol, int &iter, psFloat* fret,
	void func(psFloat *, psFloat* ), void dfunc(psFloat*, psFloat*))
{
	const int ITMAX=200;
	const psFloat EPS=1.0e-18;
	int j,its;

	func(_fp,p);
	dfunc(p,_xi);

	ncom = n;
	dcom = d;

	cublasFscal(ncom*dcom,-1,_xi,1);
	cublasFcopy(ncom*dcom,_xi, 1,_g,1); 
	cublasFcopy(ncom*dcom,_g, 1,_h,1);
	cublasFcopy(ncom*dcom,_g, 1,_xi,1);

	//for (j=0;j<n;j++) {
	//	g[j] = -xi[j];
	//	xi[j]=h[j]=g[j];
	//}

	int numGrid = (ncom + 512 -1)/512;


	hipMemset(_Finish_frprmn,0,ncom*sizeof(psFloat));

	for (its=0;its<ITMAX && cublasFasum(ncom, _Finish_frprmn,1) < ncom ;its++) {
		iter=its;
		batch_linmin(p,_xi,fret,func);
		batch_frprmn_step1<<<numGrid,512>>>( _Finish_frprmn, fret, _fp, ftol);
		if (cublasFasum(ncom,_Finish_frprmn,1) >=  ncom - 0.1)
			return;
		
		cublasFcopy(ncom,fret, 1,_fp,1); //fp=fret;
		dfunc(p,_xi);
		batch_frprmn_step2<<<numGrid,512>>>(_Finish_frprmn, _gg, _dgg, _xi,  _g);

/*		column_wise_norm2(gg, g, dcom,  ncom);
		
		dgg=gg=0.0;
		gg = cublasFnrm2(ncom, g,1);
		gg *= gg;
		dgg = cublasFdot(ncom, xi,1,xi,1) + cublasFdot(ncom, g,1,xi,1);
*/
//		for (j=0;j<n;j++) {
//			gg += g[j]*g[j];
////		  dgg += xi[j]*xi[j];
//			dgg += (xi[j]+g[j])*xi[j];
//		}

		batch_frprmn_step3<<<numGrid,512>>>(_Finish_frprmn, _xi, _g, _h, _gg, _dgg);
/*		if (gg == 0.0)
			return;
		gam=dgg/gg;


		cublasFscal(ncom*dcom,-1,xi,1);
		cublasFcopy(ncom*dcom,xi, 1,g,1);
		cublasFscal(ncom,gam,h,1);
		cublasFaxpy(ncom,1,g, 1,h,1);
		cublasFcopy(ncom,h, 1,xi,1);
*/
		//for (j=0;j<n;j++) {
		//	g[j] = -xi[j];
		//	xi[j]=h[j]=g[j]+gam*h[j];
		//}
	}

}


void batch_optim_init(int maxdim, int maxn)
{

	cublasAlloc(maxn, sizeof(psFloat), (void**)& _Finish_mnbrak);
	cublasAlloc(maxn, sizeof(psFloat), (void**)& _Finish_brent);
	cublasAlloc(maxn, sizeof(psFloat), (void**)& _Finish_frprmn);

	cublasAlloc(maxn, sizeof(int), (void**)& _Step_mnbrak);


	cublasAlloc(maxn, sizeof(psFloat), (void**)& _u);
	cublasAlloc(maxn, sizeof(psFloat), (void**)& _fu);
	cublasAlloc(maxn, sizeof(psFloat), (void**)& _x);
	cublasAlloc(maxn, sizeof(psFloat), (void**)& _fx);
	cublasAlloc(maxn, sizeof(psFloat), (void**)& _v);
	cublasAlloc(maxn, sizeof(psFloat), (void**)& _fv);
	cublasAlloc(maxn, sizeof(psFloat), (void**)& _w);
	cublasAlloc(maxn, sizeof(psFloat), (void**)& _fw);
	cublasAlloc(maxn, sizeof(psFloat), (void**)& _a);
	cublasAlloc(maxn, sizeof(psFloat), (void**)& _b);

	cublasAlloc(maxn, sizeof(psFloat), (void**)& _ax);
	cublasAlloc(maxn, sizeof(psFloat), (void**)& _fa);
	cublasAlloc(maxn, sizeof(psFloat), (void**)& _bx);
	cublasAlloc(maxn, sizeof(psFloat), (void**)& _fb);
	cublasAlloc(maxn, sizeof(psFloat), (void**)& _xx);
	cublasAlloc(maxn, sizeof(psFloat), (void**)& _fxx);
	cublasAlloc(maxn, sizeof(psFloat), (void**)& _xmin);

	cublasAlloc(maxdim*maxn, sizeof(psFloat),(void **) & _temp_dcom_ncom_1);

	cublasAlloc(maxn, sizeof(psFloat), (void**)& _gg);
	cublasAlloc(maxn, sizeof(psFloat), (void**)& _fp);
	cublasAlloc(maxn, sizeof(psFloat), (void**)& _dgg);

	cublasAlloc(maxdim*maxn, sizeof(psFloat),(void **) & _g);
	cublasAlloc(maxdim*maxn, sizeof(psFloat),(void **) & _h);
	cublasAlloc(maxdim*maxn, sizeof(psFloat),(void **) & _xi);


	cublasAlloc(maxdim*maxn, sizeof(psFloat),(void **) & xt);
	cublasAlloc(maxdim*maxn, sizeof(psFloat),(void **) & pcom_p);
	cublasAlloc(maxdim*maxn, sizeof(psFloat),(void **) & xicom_p);

}

void batch_optim_final()
{

	cublasFree( _Finish_mnbrak);
	cublasFree( _Finish_brent);
	cublasFree( _Finish_frprmn);

	cublasFree( _Step_mnbrak);


	cublasFree( _u);
	cublasFree( _fu);
	cublasFree( _x);
	cublasFree( _fx);
	cublasFree( _v);
	cublasFree( _fv);
	cublasFree( _w);
	cublasFree( _fw);
	cublasFree( _a);
	cublasFree( _b);

	cublasFree( _ax);
	cublasFree( _fa);
	cublasFree( _bx);
	cublasFree( _fb);
	cublasFree( _xx);
	cublasFree( _fxx);
	cublasFree( _xmin);

	cublasFree( _temp_dcom_ncom_1);

	cublasFree( _gg);
	cublasFree( _fp);
	cublasFree( _dgg);

	cublasFree(_g);
	cublasFree(_h);
	cublasFree(_xi);

	cublasFree(xt);
	cublasFree(pcom_p);
	cublasFree(xicom_p);

}