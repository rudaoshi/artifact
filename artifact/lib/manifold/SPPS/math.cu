#include "hip/hip_runtime.h"

#include "config.h"
#include <math.h>

#include "cumath.h"

__global__ void _gausian_kernel(  psFloat* val, psFloat*d, psFloat* d2, psFloat* x , int M, psFloat* param, CalculationType calType)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (i >= M)
		return;
	
	val[i] = exp(-x[i]*x[i]/(2*param[i]*param[i]));
	
	if(calType != Eval)
	{
		d[i] = - val[i]/(2*param[i]*param[i]);
		d2[i] = val[i]/(4*pow(param[i],4));
	
	}
}


void gausian_kernel( psFloat* val, psFloat*d, psFloat* d2, psFloat* x, int M, psFloat* param, CalculationType calType)
{
	int numGrid = (M + 512 -1)/512;
	_gausian_kernel<<<numGrid,512>>>(val, d, d2, x,M, param, calType);
}


//��X��ÿi������param[i]Ϊ������Gaussian��

__global__ void _rowwise_gausian_kernel(  psFloat* val, psFloat*d, psFloat* d2, psFloat* X , int M, int N,  psFloat* param, CalculationType calType)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	
	if (i >= M || j > N)
		return;
	
	val[i*N + j] = exp(-X[i*N + j]*X[i*N + j]/(2*param[i]*param[i]));
	
	if(calType != Eval)
	{
		d[i*N + j]  = - val[i*N + j] /(2*param[i]*param[i]);
		d2[i*N + j]  = val[i*N + j] /(4*pow(param[i],4));
	
	}
}


void rowwise_gausian_kernel( psFloat* val, psFloat*d, psFloat* d2, psFloat* X, int M, int N, psFloat* param, CalculationType calType)
{
	dim3 dimBlock(16,32);
	dim3 dimGrid((M +dimBlock.x-1)/dimBlock.x, (N+dimBlock.y-1)/dimBlock.y);
	_rowwise_gausian_kernel<<<dimGrid,dimBlock>>>(val, d, d2, X,M,N, param, calType);
}



__global__ void _quadratic_kernel( psFloat* val, psFloat*d, psFloat* d2, psFloat* x , int M, psFloat* param, CalculationType calType)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (i >= M)
		return;

	psFloat cur_val = 1 - x[i]*x[i]/(param[i]*param[i]);
	
	if (cur_val < 0) cur_val = 0;
	
	val[i]  = cur_val*cur_val;
	
	if(calType != Eval)
	{
		d[i] = - 2*cur_val/(param[i]*param[i]);
		d2[i] = 2/(pow(param[i],4));
	
	}
}



void quadratic_kernel( psFloat* val, psFloat*d, psFloat* d2, psFloat* x, int M, psFloat* param, CalculationType calType)
{
	int numGrid = (M + 512 -1)/512;
	_quadratic_kernel<<<numGrid,512>>>(val, d, d2, x,M, param, calType);
}



//��X��ÿi������param[i]Ϊ������Quadratic��

__global__ void _rowwise_quadratic_kernel(  psFloat* val, psFloat*d, psFloat* d2, psFloat* X , int M, int N,  psFloat* param, CalculationType calType)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	
	if (i >= M || j > N)
		return;
	
	psFloat cur_val = 1 - X[i*N + j]*X[i*N + j]/(param[i]*param[i]);
	
	if (cur_val < 0) cur_val = 0;
	
	val[i*N + j]  = cur_val*cur_val;
	
	if(calType != Eval)
	{
		d[i*N + j] = - 2*cur_val/(param[i]*param[i]);
		d2[i*N + j] = 2/(pow(param[i],4));
	
	}
}


void rowwise_quadratic_kernel( psFloat* val, psFloat*d, psFloat* d2, psFloat* X, int M, int N, psFloat* param, CalculationType calType)
{
	dim3 dimBlock(16,32);
	dim3 dimGrid((M +dimBlock.x-1)/dimBlock.x, (N+dimBlock.y-1)/dimBlock.y);
	_rowwise_quadratic_kernel<<<dimGrid,dimBlock>>>(val, d, d2, X,M,N, param, calType);
}

