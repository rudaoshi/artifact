#include "hip/hip_runtime.h"

#include "config.h"
#include "spps.h"

#include <math.h>
#include <hipblas.h>
#include <cutil.h>


#include "util.h"
#include "cumath.h"
#include "optim.h"


psFloat* x;
psFloat* diffVecs;
psFloat* normDiffV;
psFloat* kernelVal;
psFloat* kernelD;
psFloat* kernelD2;

psFloat* b;
psFloat* Jbx;
psFloat* Jbs;
psFloat* p;
psFloat* Jpx;
psFloat* Jps;

psFloat* q;

psFloat* Hbxx;
psFloat* Hbxs;

psFloat* mx;
psFloat* Jmx;
psFloat* Jms;

psFloat* y;
psFloat dis;
psFloat* Jdx;
psFloat* Hdxx;
psFloat* Jds;
psFloat* Hdxs;

psFloat* gx;     // ��ͶӰ����ʱ����x���й����ݶ��Ż�ʱʹ�õ�g
psFloat* hx;     // ��ͶӰ����ʱ����x���й����ݶ��Ż�ʱʹ�õ�h


psFloat* ones_M;  // Mάȫ1����
psFloat* ones_d;  // dάȫ1����
//Finished: ��������ʹ��MxM����
//psFloat* tempMxM; // �ϴ�ľ���������M x M��  
psFloat* tempD;
psFloat* tempd;
psFloat* tempM;


psFloat * mX;
psFloat * projY;


SPPS s;


void display(char * msg, psFloat * dm, int r , int c)
{
	static psFloat temp[1024*1024];
	
	for (int i = 0;i < r; i++)
	{
		for (int j = 0;j < c; j++)
		{
			temp[j*r+i]  = -1;
		}
	}
	
	hipblasGetMatrix(r,c, sizeof(psFloat), dm, r, temp,r);
	
	printf(msg); printf("\n[");
	 
	for (int i = 0;i < r; i++)
	{
		for (int j = 0;j < c; j++)
		{
			printf("%f ", temp[j*r+i]);
		}
		
		if (i < r-1)
			printf("\n");
	}
	printf("]\n");
}

void SPPS_init(psFloat* Y, int D, int N, psFloat* initS, psFloat* initT, int d, int M, KernelType kernel, psFloat* Sigma, int * nngraph, int neighbor_pair)
{
	if (cublasInit() != HIPBLAS_STATUS_SUCCESS)
		return;

	// Fill the elements of S

	s.D = D;
	s.N = N;
	s.d = d;
	s.M = M;
	s.kernel = kernel;
	s.neighbor_pair = neighbor_pair;

	cublasAlloc(s.D*s.N, sizeof(psFloat), (void **) &s.Y);
	hipblasSetMatrix(s.D, s.N, sizeof(psFloat), Y,D, s.Y, s.D);
	
//	display("Computing the SPPS for data set:", s.Y , s.D , s.N);
	
	cublasAlloc(s.d*s.N, sizeof(psFloat), (void **) &s.X);

	cublasAlloc(s.D*s.M, sizeof(psFloat), (void **) &s.T);
	hipblasSetMatrix(s.D, s.M, sizeof(psFloat),  initT,D,s.T, s.D);


	cublasAlloc(s.d*s.M, sizeof(psFloat),(void **) &s.S);
	hipblasSetMatrix(s.d, s.M, sizeof(psFloat), initS,d,  s.S, s.d);

	
	cublasAlloc(s.M, sizeof(psFloat),(void **) &s.Sigma);
	hipblasSetVector( s.M, sizeof(psFloat), Sigma, 1, s.Sigma,1);

	cublasAlloc(2*s.neighbor_pair, sizeof(int), (void **) &s.nngraph );
	hipblasSetMatrix( s.neighbor_pair,2, sizeof(int), nngraph, s.neighbor_pair ,  s.nngraph, s.neighbor_pair);

	// malloc the internal temporary memories

//	cublasAlloc(s.d, sizeof(psFloat),(void **) & x);

	cublasAlloc(s.d*s.M, sizeof(psFloat),(void **) & diffVecs);
	cublasAlloc(s.M, sizeof(psFloat),(void **) & normDiffV);
	cublasAlloc(s.M, sizeof(psFloat),(void **) & kernelVal);
	cublasAlloc(s.M, sizeof(psFloat),(void **) & kernelD);
	cublasAlloc(s.M, sizeof(psFloat),(void **) & kernelD2);


	cublasAlloc(s.M, sizeof(psFloat),(void **) & b);
	cublasAlloc(s.d*s.M, sizeof(psFloat),(void **) & Jbx);
	cublasAlloc(s.d*s.M, sizeof(psFloat),(void **) & Jbs);
	cublasAlloc(s.M, sizeof(psFloat),(void **) & p);
	cublasAlloc(s.d*s.M, sizeof(psFloat),(void **) & Jpx);
	cublasAlloc(s.d*s.M, sizeof(psFloat),(void **) & Jps);
	cublasAlloc(s.M, sizeof(psFloat),(void **) & q);

	cublasAlloc(s.d*s.d, sizeof(psFloat),(void **) & Hbxx);
	cublasAlloc(s.d*s.d, sizeof(psFloat),(void **) & Hbxs);

	cublasAlloc(s.D, sizeof(psFloat),(void **) & mx);
	cublasAlloc(s.d*s.D, sizeof(psFloat),(void **) & Jmx);
	cublasAlloc(s.d*s.D, sizeof(psFloat),(void **) & Jms);

//	cublasAlloc(s.D, sizeof(psFloat),(void **) & y);
//	cublasAlloc(s.d, sizeof(psFloat),(void **) & Jdx);
	cublasAlloc(s.d*s.d, sizeof(psFloat),(void **) & Hdxx);
	cublasAlloc(s.d, sizeof(psFloat),(void **) & Jds);
	cublasAlloc(s.d*s.d, sizeof(psFloat),(void **) & Hdxs);

	cublasAlloc(s.d, sizeof(psFloat),(void **) & gx);
	cublasAlloc(s.d, sizeof(psFloat),(void **) & hx);

	// malloc the external temporary memories

	cublasAlloc(s.M, sizeof(psFloat),(void **) & ones_M);
	fill(ones_M, s.M,1, 1.0);

	cublasAlloc(s.d, sizeof(psFloat), (void **) & ones_d);
	fill(ones_d, s.d,1, 1.0);

	cublasAlloc(s.D, sizeof(psFloat),(void **) & tempD);
	cublasAlloc(s.d, sizeof(psFloat),(void **) & tempd);
	cublasAlloc(s.M, sizeof(psFloat),(void **) & tempM);

	cublasAlloc(s.D*s.M, sizeof(psFloat), (void **) &mX);
	cublasAlloc(s.D*s.M, sizeof(psFloat), (void **) &projY);
	
	optim_init(max(s.D,s.M));

}


void SPPS_final()
{

	// Free the elements of SPPS
	cublasFree(s.Y);

	cublasFree(s.T);

	cublasFree(s.S);
	
	cublasFree(s.Sigma);

	cublasFree(s.nngraph );

	// free the internal temporary memories

//	cublasFree( x);
	cublasFree( diffVecs);
	cublasFree( normDiffV);
	cublasFree( kernelVal);
	cublasFree( kernelD);
	cublasFree( kernelD2);


	cublasFree( b);
	cublasFree( Jbx);
	cublasFree( Jbs);
	cublasFree( p);
	cublasFree( Jpx);
	cublasFree( Jps);
	cublasFree( q);

	cublasFree( Hbxx);
	cublasFree( Hbxs);

	cublasFree( mx);
	cublasFree( Jmx);
	cublasFree( Jms);

//	cublasFree( y);
	cublasFree( Jdx);
	cublasFree( Hdxx);
	cublasFree( Jds);
	cublasFree( Hdxs);

	cublasFree( gx);
	cublasFree( hx);

	// free the external temporary memories

	cublasFree( ones_M);
	cublasFree( ones_d);

	cublasFree(  tempD);
	cublasFree(  tempd);
	cublasFree(  tempM);
	cublasFree(  mX);
	cublasFree(  projY);

	
	optim_final();


}

void slps_regkernel( CalculationType calType)
{

	column_wise_norm2(normDiffV, diffVecs, s.d, s.M);
	
#ifdef _DEBUG_DETAIL
	display("the resulting normDiffV matrix is:", normDiffV, 1 , s.M);
#endif

	if (s.kernel == Gaussian)
	{
		gausian_kernel( kernelVal, kernelD, kernelD2, normDiffV, s.M, s.Sigma, calType);

	}
	else if(s.kernel == Quadratic)
	{
		quadratic_kernel( kernelVal, kernelD, kernelD2, normDiffV, s.M, s.Sigma, calType);

	}

#ifdef _DEBUG_DETAIL
	display("the resulting kernelVal matrix is:", kernelVal, 1 , s.M);
#endif
	
}




void slps_kernelregbasis(psFloat * _x, CalculationType calType)
{
//	psFloat* x = x;

	// calculate the diffVecs
#ifdef _DEBUG_DETAIL
	display("the S matrix is:", s.S, s.d , s.M);
#endif
	column_wise_add(diffVecs, 1,  s.S, s.d, s.M, -1, _x);
	//cublasFcopy(s.d * s.M, s.S,  1, diffVecs, 1);
	//cublasFger (s.d, s.M, 1.0, x, 1, ones_M, 1, diffVecs, s.d);

#ifdef _DEBUG_DETAIL
	display("the resulting diffVecs matrix is:", diffVecs, s.d , s.M);
#endif
	
	
	// calculate the kernel value and kernel differences
	slps_regkernel(calType);
	
	// calculate b
	psFloat sumval = cublasFasum(s.M, kernelVal, 1);
	
	if (sumval < 1e-6)
	{ 
		sumval = 1;
	}
	
	cublasFcopy (s.M, kernelVal,  1, b, 1);
	cublasFscal (s.M, 1/sumval, b, 1);
	
#ifdef _DEBUG_DETAIL
	display("the resulting b matrix is:", b, 1 , s.M);
#endif
	
	if (calType == Eval)
		return;
			
	// Calculate p
	cublasFcopy (s.M, kernelD,  1, p, 1);
	cublasFscal (s.M, -2/sumval, p, 1);
#ifdef _DEBUG_DETAIL
	display("the resulting p matrix is:", p, 1 , s.M); 
#endif
	// Calculate q
	cublasFcopy (s.M, kernelD2,  1, q, 1);
	cublasFscal (s.M, 4/sumval, q, 1);
#ifdef _DEBUG_DETAIL
	display("the resulting q matrix is:", q, 1 , s.M); 
#endif
	// Calculate Jbx
	// Jbx = diffVecs*(diag(p)-p*b');
	// 1. Jbx = diffVecs*diag(p);
	column_wise_scal(Jbx,diffVecs, s.d, s.M, p);
	// 2. tempd = diffVecs*p;
	cublasFgemv('N', s.d,s.M, 1, diffVecs, s.d, p, 1, 0, tempd,1);
	// 3. Jbx = Jbx - tempd*b'
	cublasFger(s.d,s.M, -1, tempd, 1, b, 1, Jbx, s.d);
#ifdef _DEBUG_DETAIL
	display("the resulting Jbx matrix is:", Jbx, s.d , s.M);
#endif
			
	// Calculate Jpx
	// Jpx =  diffVecs*(diag(q) - p*p');
	// 1. Jpx = diffVecs*diag(q);
	column_wise_scal(Jpx,diffVecs, s.d, s.M, q);
	// 2. tempd = diffVecs*p; has already been calculated.
	// cublasFgemv('N', s.d,s.M, 1, diffVecs, s.d, p, 1, 0, tempd,1);
	// 3. Jpx = Jpx - tempd*p'
	cublasFger(s.d,s.M, -1, tempd, 1, p, 1, Jpx, s.d);

}

void slps_map(psFloat * _mx, psFloat * _Jmx, psFloat * _x,  CalculationType calType)
{

	slps_kernelregbasis(_x, calType);
	// Compute mx
	// ps.mx = ps.T*ps.b;
	cublasFgemv('N', s.D,s.M,1,  s.T, s.D, 
			b, 1, 0, _mx, 1);
			
			
	if (calType == Eval )
		return;
	
	// Compute Jmx
	// ps.Jmx = ps.Jbx*ps.T';
	
	cublasFgemm('N','T',s.d,s.D,s.M, 1, Jbx, s.d, 
			s.T, s.D, 0, _Jmx, s.d);
	
}

// Compute the distance 
void slps_dist(psFloat * dist, psFloat * _mx, psFloat * _Jmx, psFloat * _Jdx, psFloat * _Hdxx, psFloat * _x, psFloat * _y, CalculationType calType)
{
	slps_map( _mx, _Jmx, _x, calType);
	
	// tempD = y - ps.mx;
	cublasFcopy(s.D, _y, 1,  tempD, 1);
	cublasFaxpy(s.D, -1, _mx, 1, tempD, 1);.
	
	// dist = norm(tempD);
	*dist = cublasFnrm2(s.D, tempD, 1);
	*dist *= *dist;
	
	if (calType == Eval)
		return;
	
	//  Jdx = 2*ps.Jmx*(ps.mx-y);
#ifdef _DEBUG_DETAIL
	display("the resulting Jmx matrix is:", _Jmx, s.d , s.D);
	display("the resulting y-mx matrix is:", tempD, 1 , s.D);
#endif
	cublasFgemv('N', s.d, s.D, -2, _Jmx, s.d,  tempD, 1, 1, _Jdx,1);
	
	if (calType == Jacobbi)
		return;
	
	// Calculate Hdxx
	psFloat sumP = cublasFasum(s.M, p, 1);
	
	hipMemset(_Hdxx,0,s.d*s.d*sizeof(psFloat));

	for (int i = 0; i < s.M; i++)
	{
		//Calculate Hbxx(:,:,i);
		//(b(i)*sum(p)-p(i))*eye(d) + diffVecs(:,i)*Jpx(:,i)' - diffVecs*p*Jbx(:,i)' - b(i)*diffVecs*Jpx';
		
		hipMemset(Hbxx,0,s.d*s.d*sizeof(psFloat));
		//Hbxx = (b(i)*sum(p)-p(i))*eye(d)
		fill(Hbxx,s.d*s.d, s.d, b[i]*sumP-p[i]);
		//Hbxx += diffVecs(:,i)*Jpx(:,i)'
		cublasFger(s.d,s.d, 1, diffVecs + i*s.d, 1, Jpx + i*s.d, 1, Hbxx, s.d);
		
		//Hbxx +=  - diffVecs*p*Jbx(:,i)'
		cublasFgemv('N', s.d,s.M, 1, diffVecs, s.d, p, 1, 0, tempd,1);
		cublasFger(s.d,s.d, -1, tempD, 1, Jbx + i*s.d, 1, Hbxx, s.d);
		
		//Hbxx +=  - b(i)*diffVecs*Jpx'
		cublasFgemm('N','T',s.d,s.M,s.d, -b[i], diffVecs, s.d, 
			Jpx, s.d, 0, Hbxx, s.d);

		// Hdxx = Hdxx + 2*dot(ps.T(:,i),ps.mx - y)*ps.Hbxx(:,:,i) + 2*ps.Jbx(:,i)*ps.T(:,i)'*ps.T*ps.Jbx';
		psFloat coe = - cublasFdot(s.D, s.T+ i*s.D, 1, tempD, 1);
		// Hdxx += 2*dot(ps.T(:,i),ps.mx - y)*ps.Hbxx(:,:,i)
		cublasFaxpy(s.d*s.d, 2*coe, Hbxx, 1, _Hdxx,1);
		
		// tempM = ps.T'*ps.T(:,i)
		cublasFgemv('T', s.M,s.D, 1, s.T, s.D, s.T + i*s.D, 1, 0, tempM,1);
		// tempd = ps.Jbx*tempM = ps.Jbx*ps.T'*ps.T(:,i)
		cublasFgemv('N', s.d,s.M, 1, Jbx, s.d, tempM, 1, 0, tempd,1);
		// Hdxx += 2*ps.Jbx(:,i)*ps.T(:,i)'*ps.T*ps.Jbx' = 2*ps.Jbx(:,i)*tempd';
		cublasFger(s.d,s.d, 2, Jbx + i*s.d, 1, tempd, 1, _Hdxx, s.d);

	}	
}


// �Ż���������,������ĳ��y���������ͶӰָ��
// �������������ĳ��y��S(x)�ľ���
// ʹ��ǰ���뽫y����������y�С�
psFloat slps_dist_help(psFloat* _x )
{

//	cublasFcopy(s.d, x, 1, x,1);


	slps_dist(& dis, mx, Jmx, Jdx, Hdxx, _x, y,Eval);

//	static int i = 0;
	
//	i ++;
	
//	if (i % 10 == 0)
#ifdef _DEBUG_DETAIL
    display("computing the object value at point:", _x, 1 , s.d);
#endif

	return dis;
}

// �Ż���������,������ĳ��y���������ͶӰָ��
// �������������ĳ��y��S(x)�ľ����x�ĵ���
// ʹ��ǰ���뽫y����������y�У�����slps_dist_help��֮ǰ�����ű����ù���
void slps_dist_Jacobbi_help(psFloat* _x ,psFloat*  _Jdx )
{

	slps_dist(& dis, mx, Jmx, _Jdx, Hdxx, _x, y,Jacobbi) ;

#ifdef _DEBUG_DETAIL	
	display("computing the Jacobbi matrix at point:", _x, 1 , s.d);
	display("the resulting Jacobbi matrix is:", _Jdx, 1 , s.d);
#endif
	
//	cublasFcopy(s.d, Jdx, 1, Jdx,1);
}

void slps_project(psFloat * _x, psFloat* _y, psFloat* x0, bool withInitVal)
{
	/**
	* Ѱ�ҵ���Ӧ�ĳ�ʼ��x
	*/
	
//	x = x0;
	y = _y;
	
	if (!withInitVal)
	{
		column_wise_distance(tempM, s.T, s.D, s.M,y);
//		display("the pairwise distance are:", tempM, 1 , s.M);

		// Ѱ��tempM����Сֵ��λ�á�
		int index = cublasIfamin(s.M,tempM,1);
		index --;
		cublasFcopy(s.d, s.S + index*s.d, 1, _x,1);
	}
	else
	{
		cublasFcopy(s.d, x0, 1, _x,1);
	}
#ifdef _DEBUG_DETAIL	
	display("The initial point is:", _x, 1 , s.d);
#endif
	int iter;
	psFloat fret;
	psFloat ftol = 1e-6;
	
	// the resulting x is the projection index.
	frprmn(s.d, _x, ftol, iter, fret, &slps_dist_help,&slps_dist_Jacobbi_help);

}

void slps_map(psFloat * Y, psFloat * _X,  int N)
{
	for ( int i = 0; i <  N; i++)
	{
#ifdef _DEBUG_DETAIL	
		display("Calculate the mapping for the point:",  _X + s.d*i, 1 , s.d);
#endif
		slps_map( Y + s.D*i, Jmx, _X + s.d*i, Eval);
		
	}
}

void slps_project(psFloat * _X, psFloat * Y, int N)
{
	for ( int i = 0; i < N; i++)
	{
//		printf("Computing the projection of %d-th sample\n", i);
		//if (i == 100)
		//{
		//	bool stop;
		//	stop = true;
		//}
		slps_project(  _X + s.d*i, Y + s.D*i,0 , false);
		
	}
}


void slps_map_train(psFloat * _mX)
{
	for ( int i = 0; i < s.M; i++)
	{
#ifdef _DEBUG_DETAIL	
		display("Calculate the mapping for the point:",  s.S + s.d*i, 1 , s.d);
#endif
		slps_map( mX + s.D*i, Jmx,s.S + s.d*i, Eval);
	}

	hipblasGetMatrix(s.D, s.M, sizeof(psFloat), mX, s.D, _mX, s.D);
}


void slps_project_train(psFloat * _X )
{
	for ( int i = 0; i < s.N; i++)
	{
//		printf("Computing the projection of %d-th sample\n", i);
		//if (i == 100)
		//{
		//	bool stop;
		//	stop = true;
		//}
		slps_project(projY + s.d*i,  s.Y + s.D*i,0 , false);
		
	}
	hipblasGetMatrix(s.d, s.N, sizeof(psFloat), projY, s.d, _X, s.d);
}


//
//void slps_get_image(psFloat * image)
//{
//	hipblasGetMatrix(s.D, s.N, sizeof(psFloat), s.X, s.d, trainfeature, s.d);
//}
//
//void slps_get_trainfeature(psFloat * trainfeature)
//{
//	hipblasGetMatrix(s.d, s.N, sizeof(psFloat), s.X, s.d, trainfeature, s.d);
//}