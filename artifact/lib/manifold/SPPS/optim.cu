#include "hip/hip_runtime.h"

#include "config.h"
#include <math.h>
#include <hipblas.h>
#include <float.h>



void swap(psFloat& a, psFloat& b)
{
	psFloat c = b;
	b = a;
	a = c;
}


void shft3(psFloat&a, psFloat&b, psFloat&c, const psFloat d)
{
	a=b;
	b=c;
	c=d;
}

psFloat sign(const psFloat&a, const psFloat&b)
{
	return b >= 0 ? (a >= 0 ? a : -a) : (a >= 0 ? -a : a);
}


void mnbrak(psFloat &ax, psFloat &bx, psFloat &cx, psFloat &fa, psFloat &fb, psFloat &fc,
	psFloat func(const psFloat))
{
	const psFloat GOLD=1.618034,GLIMIT=100.0,TINY=1.0e-20;
	psFloat ulim,u,r,q,fu;

	fa=func(ax);
	fb=func(bx);
	if (fb > fa) {
		swap(ax,bx);
		swap(fb,fa);
	}
	cx=bx+GOLD*(bx-ax);
	fc=func(cx);
	while (fb > fc) {
		r=(bx-ax)*(fb-fc);
		q=(bx-cx)*(fb-fa);
		u=bx-((bx-cx)*q-(bx-ax)*r)/
			(2.0*sign(max(fabs(q-r),TINY),q-r));
		ulim=bx+GLIMIT*(cx-bx);
		if ((bx-u)*(u-cx) > 0.0) {
			fu=func(u);
			if (fu < fc) {
				ax=bx;
				bx=u;
				fa=fb;
				fb=fu;
				return;
			} else if (fu > fb) {
				cx=u;
				fc=fu;
				return;
			}
			u=cx+GOLD*(cx-bx);
			fu=func(u);
		} else if ((cx-u)*(u-ulim) > 0.0) {
			fu=func(u);
			if (fu < fc) {
				shft3(bx,cx,u,cx+GOLD*(cx-bx));
				shft3(fb,fc,fu,func(u));
			}
		} else if ((u-ulim)*(ulim-cx) >= 0.0) {
			u=ulim;
			fu=func(u);
		} else {
			u=cx+GOLD*(cx-bx);
			fu=func(u);
		}
		shft3(ax,bx,cx,u);
		shft3(fa,fb,fc,fu);
	}
}

psFloat brent(const psFloat ax, const psFloat bx, const psFloat cx, psFloat f(const psFloat),
	const psFloat tol, psFloat&xmin)
{
	const int ITMAX=100;
	const psFloat CGOLD=0.3819660;
	const psFloat ZEPS=DBL_EPSILON*1.0e-3;
	int iter;
	psFloat a,b,d=0.0,etemp,fu,fv,fw,fx;
	psFloat p,q,r,tol1,tol2,u,v,w,x,xm;
	psFloat e=0.0;

	a=(ax < cx ? ax : cx);
	b=(ax > cx ? ax : cx);
	x=w=v=bx;
	fw=fv=fx=f(x);
	for (iter=0;iter<ITMAX;iter++) {
		xm=0.5*(a+b);
		tol2=2.0*(tol1=tol*fabs(x)+ZEPS);
		if (fabs(x-xm) <= (tol2-0.5*(b-a))) {
			xmin=x;
			return fx;
		}
		if (fabs(e) > tol1) {
			r=(x-w)*(fx-fv);
			q=(x-v)*(fx-fw);
			p=(x-v)*q-(x-w)*r;
			q=2.0*(q-r);
			if (q > 0.0) p = -p;
			q=fabs(q);
			etemp=e;
			e=d;
			if (fabs(p) >= fabs(0.5*q*etemp) || p <= q*(a-x) || p >= q*(b-x))
				d=CGOLD*(e=(x >= xm ? a-x : b-x));
			else {
				d=p/q;
				u=x+d;
				if (u-a < tol2 || b-u < tol2)
					d=sign(tol1,xm-x);
			}
		} else {
			d=CGOLD*(e=(x >= xm ? a-x : b-x));
		}
		u=(fabs(d) >= tol1 ? x+d : x+sign(tol1,d));
		fu=f(u);
		if (fu <= fx) {
			if (u >= x) a=x; else b=x;
			shft3(v,w,x,u);
			shft3(fv,fw,fx,fu);
		} else {
			if (u < x) a=u; else b=u;
			if (fu <= fw || w == x) {
				v=w;
				w=u;
				fv=fw;
				fw=fu;
			} else if (fu <= fv || v == x || v == w) {
				v=u;
				fv=fu;
			}
		}
	}
//	nrerror("Too many iterations in brent");
	xmin=x;
	return fx;
}

/**
* 
*/

int ncom;
psFloat(*nrfunc)(psFloat*);

psFloat * xt;


psFloat *pcom_p;
psFloat *xicom_p;

psFloat f1dim( psFloat x)
{
	int j;

	psFloat* pcom = pcom_p, * xicom = xicom_p;

	cublasFcopy(ncom,pcom, 1,xt,1);
	cublasFaxpy(ncom,x,xicom, 1,xt,1);

	//for (j=0;j<ncom;j++)
	//	xt[j]=pcom[j]+x*xicom[j];
	return nrfunc(xt);
}



void linmin(psFloat* p, psFloat* xi, psFloat&fret, psFloat func(psFloat*))
{
	int j;
	const psFloat TOL=1.0e-8;
	psFloat xx,xmin,fx,fb,fa,bx,ax;


	nrfunc=func;

	cublasFcopy(ncom,p, 1,pcom_p,1);
	cublasFcopy(ncom,xi, 1,xicom_p,1);
	//Vec_DP &pcom=*pcom_p,&xicom=*xicom_p;
	//for (j=0;j<n;j++) {
	//	pcom[j]=p[j];
	//	xicom[j]=xi[j];
	//}
	ax=0.0;
	xx=1.0;
	mnbrak(ax,xx,bx,fa,fx,fb,f1dim);
	fret=brent(ax,xx,bx,f1dim,TOL,xmin);

	cublasFscal(ncom, xmin,xi,1);
	cublasFaxpy(ncom,1,xi,1,p,1);
	//for (j=0;j<n;j++) {
	//	xi[j] *= xmin;
	//	p[j] += xi[j];
	//}
	//delete xicom_p;
	//delete pcom_p;
}




psFloat* g;
psFloat* h;
psFloat* xi;

/**
* �����ݶȷ��������Ż���
*/

void frprmn(int n, psFloat* p, const psFloat ftol, int &iter, psFloat&fret,
	psFloat func(psFloat* ), void dfunc(psFloat*, psFloat*))
{
	const int ITMAX=200;
	const psFloat EPS=1.0e-18;
	int j,its;
	psFloat gg,gam,fp,dgg;

	fp=func(p);
	dfunc(p,xi);

	ncom = n;

	cublasFscal(ncom,-1,xi,1);
	cublasFcopy(ncom,xi, 1,g,1); 
	cublasFcopy(ncom,g, 1,h,1);
	cublasFcopy(ncom,g, 1,xi,1);

	//for (j=0;j<n;j++) {
	//	g[j] = -xi[j];
	//	xi[j]=h[j]=g[j];
	//}

	for (its=0;its<ITMAX;its++) {
		iter=its;
		linmin(p,xi,fret,func);
		if (2.0*fabs(fret-fp) <= ftol*(fabs(fret)+fabs(fp)+EPS))
			return;
		fp=fret;
		dfunc(p,xi);
		dgg=gg=0.0;
		gg = cublasFnrm2(ncom, g,1);
		gg *= gg;
		dgg = cublasFdot(ncom, xi,1,xi,1) + cublasFdot(ncom, g,1,xi,1);

//		for (j=0;j<n;j++) {
//			gg += g[j]*g[j];
////		  dgg += xi[j]*xi[j];
//			dgg += (xi[j]+g[j])*xi[j];
//		}
		if (gg == 0.0)
			return;
		gam=dgg/gg;


		cublasFscal(ncom,-1,xi,1);
		cublasFcopy(ncom,xi, 1,g,1);
		cublasFscal(ncom,gam,h,1);
		cublasFaxpy(ncom,1,g, 1,h,1);
		cublasFcopy(ncom,h, 1,xi,1);

		//for (j=0;j<n;j++) {
		//	g[j] = -xi[j];
		//	xi[j]=h[j]=g[j]+gam*h[j];
		//}
	}

}


void optim_init(int maxdim)
{

	cublasAlloc(maxdim, sizeof(psFloat),(void **) & g);
	cublasAlloc(maxdim, sizeof(psFloat),(void **) & h);
	cublasAlloc(maxdim, sizeof(psFloat),(void **) & xi);
	cublasAlloc(maxdim, sizeof(psFloat),(void **) & xt);

	cublasAlloc(maxdim, sizeof(psFloat),(void **) & pcom_p);
	cublasAlloc(maxdim, sizeof(psFloat),(void **) & xicom_p);

}

void optim_final()
{
	cublasFree(g);
	cublasFree(h);
	cublasFree(xi);
	cublasFree(xt);

	cublasFree(pcom_p);
	cublasFree(xicom_p);

}